#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <vector>
#include <string>
#include <algorithm>
#include <hip/hip_runtime.h>
#include "json.hpp"
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/copy.h>
#include <chrono>

#define VERTEX_NUMBER 10
#define COMBINATION_NUMBER 1013
#define THREADS_NUMBER 10

using json = nlohmann::json;


__constant__ int const_adjacency_matrix[VERTEX_NUMBER][VERTEX_NUMBER];
__device__ int const_combinations_array[COMBINATION_NUMBER][VERTEX_NUMBER];


    // ADJENCY MATRIX SIZE 
__global__ void myKernel(int threads_number, int combination_number, int vertex_number, int* result_array) {
    int id_x = threadIdx.x;

    for(int c = id_x; c < combination_number; c+=threads_number) {
        bool isClique = true;
        for(int i = 0; i < vertex_number; i++) {
            if(const_combinations_array[c][i] == -1) {
                break;
            }
            for(int j = 0; j < vertex_number; j++) {
                if(const_combinations_array[c][j] == -1) {
                    break;
                }
                if(const_combinations_array[c][i] == const_combinations_array[c][j]) {
                    continue;
                }
                if(const_adjacency_matrix[const_combinations_array[c][i]][const_combinations_array[c][j]] != 1) {
                    isClique = false;
                }
            }
        }
        if(isClique) {
            result_array[c] = 1;
        } else {
            result_array[c] = 0;
        }
    }
    
}

int main() {
    std::ifstream file("combinations.json");
    json jsonData;
    file >> jsonData;

    //                                COMBINATION DATA 

    int combinations_host_array[COMBINATION_NUMBER][VERTEX_NUMBER];
    for (int i = 0; i < COMBINATION_NUMBER; ++i) {
        for (int j = 0; j < VERTEX_NUMBER; ++j) {
            combinations_host_array[i][j] = -1;
        }
    }

    int index = 0;
    for (const auto& item : jsonData) {
        if (index >= COMBINATION_NUMBER) {
            break;
        }
        
        auto vec = item.get<std::vector<int>>();
        for (size_t i = 0; i < vec.size(); ++i) {
            if (i >= VERTEX_NUMBER) {
                break;
            }
            combinations_host_array[index][i] = vec[i];
        }
        ++index;
    }

    //CHANGE VERTEXES SIZE
    int adjacency_matrix[VERTEX_NUMBER][VERTEX_NUMBER] = {
        {0, 1, 0, 0, 1, 0, 0, 0, 0, 0},
        {1, 0, 0, 0, 1, 0, 0, 0, 0, 0},
        {0, 0, 0, 0, 0, 0, 0, 0, 0, 0},
        {0, 0, 0, 0, 0, 0, 0, 0, 0, 0},
        {1, 1, 0, 0, 0, 0, 0, 0, 0, 0},
        {0, 0, 0, 0, 0, 0, 0, 0, 0, 0},
        {0, 0, 0, 0, 0, 0, 0, 0, 0, 0},
        {0, 0, 0, 0, 0, 0, 0, 0, 0, 0},
        {0, 0, 0, 0, 0, 0, 0, 0, 0, 0},
        {0, 0, 0, 0, 0, 0, 0, 0, 0, 0}
    };

    // COPY ALL DATA TO GPU
    hipMemcpyToSymbol(HIP_SYMBOL(const_adjacency_matrix), adjacency_matrix, VERTEX_NUMBER * VERTEX_NUMBER * sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(const_combinations_array), combinations_host_array, COMBINATION_NUMBER * VERTEX_NUMBER * sizeof(int));

    //RESULT_ARRAY
    int* h_result_array = (int*)malloc(COMBINATION_NUMBER * sizeof(int));
    for (int i = 0; i < COMBINATION_NUMBER; ++i) {
        h_result_array[i] = -1;
    }

    int* d_result_array;
    hipMalloc(&d_result_array, COMBINATION_NUMBER * sizeof(int));
    hipMemcpy(d_result_array, h_result_array, COMBINATION_NUMBER * sizeof(int), hipMemcpyHostToDevice);

    std::cout << "BEFORE: \n";
    auto start = std::chrono::steady_clock::now();
    myKernel<<<1, THREADS_NUMBER>>>(THREADS_NUMBER, COMBINATION_NUMBER, VERTEX_NUMBER, d_result_array);
    auto end = std::chrono::steady_clock::now();;
    auto duration = std::chrono::duration_cast<std::chrono::nanoseconds>(end - start);

    std::cout << "Time elapsed: " << duration.count() << " nanoseconds" << std::endl;

    hipMemcpy(h_result_array, d_result_array, COMBINATION_NUMBER * sizeof(int), hipMemcpyDeviceToHost);
    std::cout << "RESULT: \n";
    for (int i = 0; i < COMBINATION_NUMBER; ++i) {
        if( h_result_array[i] == 1) {
            for (int j = 0; j < VERTEX_NUMBER; ++j) {
                std::cout << combinations_host_array[i][j] << " ";
            }
            std::cout << std::endl;
        }
    }
    std::cout << std::endl;

    hipFree(d_result_array);
    free(h_result_array);
    return 0;
}